#include "hip/hip_runtime.h"
#include <algorithm>
#include <thread>
#include <External/AnyOption/anyoption.h>
#include <Utils/Chrono.cuh>
#include "DP/VRPModel.cuh"
#include "DP/CTWModel.cuh"
#include "OffloadBuffer.cuh"
#include "BB/PriorityQueue.cuh"
#include "Options.h"

using namespace std;
using namespace Memory;
using namespace Chrono;
using namespace BB;
using namespace DD;
using namespace DP;
using namespace OP;
using ProblemType = VRProblem;
using StateType = VRPState;

// Auxiliary functions
AnyOption* parseOptions(int argc, char* argv[]);

void configGPU();

// Search
template<typename ProblemType, typename StateType>
void updatePriorityQueue(StateType* bestSolution, unsigned int * filteredStatesCount, PriorityQueue<StateType>* priorityQueue, OffloadBuffer<ProblemType, StateType>* offloadBuffer);

template<typename StateType>
bool boundsCheck(StateType* bestSolution, AugmentedState<StateType> const * augmentedState);

template<typename ProblemType, typename StateType>
bool checkForBetterSolutions(StateType* bestSolution, StateType* currentSolution, OffloadBuffer<ProblemType,StateType>* offloadBuffer);

// Offload
template<typename ProblemType, typename StateType>
void prepareOffload(StateType* bestSolution, unsigned int* filteredStatesCount, PriorityQueue<StateType>* priorityQueue, OffloadBuffer<ProblemType, StateType>* offloadBuffer);

template<typename ProblemType, typename StateType>
void prepareOffload(AugmentedState<StateType> const * augmentedState, OffloadBuffer<ProblemType,StateType>* offloadBuffer);

template<typename ProblemType, typename StateType>
void doOffloadCpuAsync(OffloadBuffer<ProblemType,StateType>* cpuOffloadBuffer, Vector<std::thread>* cpuThreads, bool onlyRestricted);

template<typename ProblemType, typename StateType>
void doOffloadGpuAsync(OffloadBuffer<ProblemType,StateType>* gpuOffloadBuffer, bool onlyRestricted);

template<typename ProblemType, typename StateType>
void doOffloadsAsync(OffloadBuffer<ProblemType,StateType>* cpuOffloadBuffer, Vector<std::thread>* cpuThreads, OffloadBuffer<ProblemType,StateType>* gpuOffloadBuffer, bool onlyRestricted);

template<typename ProblemType, typename StateType>
__global__ void doOffloadKernel(OffloadBuffer<ProblemType,StateType>* offloadBuffer, bool onlyRestricted);

void waitOffloadCpu(Vector<std::thread>* cpuThreads, uint64_t* cpuOffloadEndTime);

void waitOffloadGpu(uint64_t* gpuOffloadEndTime);

void waitOffloads(Vector<std::thread>* cpuThreads, uint64_t* cpuOffloadEndTime, uint64_t* gpuOffloadEndTime);

// Debug
void printElapsedTime(uint64_t elapsedTimeMs);

void clearLine();

int main(int argc, char* argv[])
{
    // Input parsing
    Options options;
    if (not options.parseOptions(argc, argv))
    {
        return EXIT_FAILURE;
    }
    else
    {
        options.printOptions();
    }

    // *******************
    // Data initialization
    // *******************

    // Context initialization
    std::mt19937 rng(options.randomSeed);
    MallocType gpuMallocType = MallocType::Std;
    if (options.parallelismGpu > 0)
    {
        gpuMallocType = MallocType::Managed;
        configGPU();
    };
    Vector<std::thread>* cpuThreads = new Vector<std::thread>(options.parallelismCpu, MallocType::Std);

    // Problems
    ProblemType* const cpuProblem = parseInstance<ProblemType>(options.inputFilename, MallocType::Std);
    ProblemType* const gpuProblem = parseInstance<ProblemType>(options.inputFilename, gpuMallocType);

    // PriorityQueue
    PriorityQueue<StateType> priorityQueue(cpuProblem, options.queueSize);

    // Offload
    unsigned int memorySize = sizeof(OffloadBuffer<ProblemType,StateType>);
    byte* memory = safeMalloc(memorySize, MallocType::Std);
    OffloadBuffer<ProblemType,StateType>* cpuOffloadBuffer = new (memory) OffloadBuffer<ProblemType,StateType>(cpuProblem, options.widthCpu, options.parallelismCpu, MallocType::Std);
    memory = safeMalloc(memorySize, gpuMallocType);
    OffloadBuffer<ProblemType,StateType>* gpuOffloadBuffer = new (memory) OffloadBuffer<ProblemType,StateType>(gpuProblem, options.widthGpu, options.parallelismGpu, gpuMallocType);

    // Solutions
    memorySize = sizeof(StateType);
    memory = safeMalloc(memorySize, MallocType::Std);
    StateType* bestSolution = new (memory) StateType(cpuProblem, MallocType::Std);
    bestSolution->cost = DP::MaxCost;
    memory = safeMalloc(memorySize, MallocType::Std);
    StateType* currentSolution = new (memory) StateType(cpuProblem, MallocType::Std);
    currentSolution->cost = DP::MaxCost;

    // Root
    memorySize = sizeof(StateType);
    memory = safeMalloc(memorySize, MallocType::Std);
    StateType* root = new (memory) StateType(cpuProblem, MallocType::Std);
    makeRoot(cpuProblem, root);
    AugmentedState<StateType> const augmentedRoot(DP::MaxCost, 0, root);
    priorityQueue.insert(&augmentedRoot);

    // Search
    unsigned int iterationsCount = 0;
    enum SearchStatus {BB, LNS} searchStatus = SearchStatus::BB;
    unsigned int visitedStatesCount = 0;
    unsigned int filteredStatesCount = 0;

    // ************
    // Begin search
    // ************
    clearLine();
    printf("[INFO] Start branch and bound search\n");
    uint64_t searchStartTime = now();
    do
    {
        switch(searchStatus)
        {
            case SearchStatus::BB:
            {
                if (priorityQueue.isFull())
                {
                    searchStatus = SearchStatus::LNS;
                    clearLine();
                    printf("[INFO] Switching to large neighborhood search\n");
                }

                prepareOffload(bestSolution, &filteredStatesCount, &priorityQueue, cpuOffloadBuffer);
                prepareOffload(bestSolution, &filteredStatesCount, &priorityQueue, gpuOffloadBuffer);
            }
                break;
            case SearchStatus::LNS:
            {
                prepareOffload(&augmentedRoot, cpuOffloadBuffer);
                prepareOffload(&augmentedRoot, gpuOffloadBuffer);
                cpuOffloadBuffer->generateNeighbourhoods(currentSolution, options.eqProbability, options.neqProbability, &rng);
                gpuOffloadBuffer->generateNeighbourhoods(currentSolution, options.eqProbability, options.neqProbability, &rng);
                currentSolution->makeInvalid();
            }
                break;
        }

        uint64_t cpuOffloadStartTime = now();
        uint64_t gpuOffloadStartTime = now();
        doOffloadsAsync(cpuOffloadBuffer, cpuThreads, gpuOffloadBuffer, searchStatus == SearchStatus::LNS);

        uint64_t cpuOffloadEndTime;
        uint64_t gpuOffloadEndTime;
        waitOffloads(cpuThreads, &cpuOffloadEndTime, &gpuOffloadEndTime);

        visitedStatesCount += cpuOffloadBuffer->getSize();
        visitedStatesCount += gpuOffloadBuffer->getSize();

        bool foundBetterSolution =
                checkForBetterSolutions(bestSolution, currentSolution, cpuOffloadBuffer) or
                checkForBetterSolutions(bestSolution, currentSolution, gpuOffloadBuffer);

        updatePriorityQueue(bestSolution, &filteredStatesCount, &priorityQueue, cpuOffloadBuffer);
        updatePriorityQueue(bestSolution, &filteredStatesCount, &priorityQueue, gpuOffloadBuffer);

        if(foundBetterSolution)
        {
            clearLine();
            printf("[INFO] Better solution found: ");
            bestSolution->selectedValues.print(false);
            printf(" | Value: %u", bestSolution->cost);
            printf(" | Time: ");
            printElapsedTime(now() - searchStartTime);
            printf(" | Iterations: %u", iterationsCount);
            printf(" | States: %u - %u - %u\n", visitedStatesCount, priorityQueue.getSize(), filteredStatesCount);
        }
        else
        {
            unsigned long int cpuSpeed = 0;
            if (cpuOffloadBuffer->getSize() > 0)
            {
                uint64_t cpuOffloadElapsedTime = max(1ul, cpuOffloadEndTime - cpuOffloadStartTime);
                cpuSpeed = cpuOffloadBuffer->getSize() * 1000 / cpuOffloadElapsedTime;
            }

            unsigned long int gpuSpeed = 0;
            if (gpuOffloadBuffer->getSize() > 0)
            {
                uint64_t gpuOffloadElapsedTime = max(1ul, gpuOffloadEndTime - gpuOffloadStartTime);
                gpuSpeed = gpuOffloadBuffer->getSize() * 1000 / gpuOffloadElapsedTime;
            }
            printf("[INFO] Solution: ");
            currentSolution->selectedValues.print(false);
            printf(" | Value: %u", currentSolution->cost);
            printf(" | Time: ");
            printElapsedTime(now() - searchStartTime);
            printf(" | Iteration: %u", iterationsCount);
            printf(" | States: %u - %u - %u", visitedStatesCount, priorityQueue.getSize(), filteredStatesCount);
            printf(" | Speed: %lu - %lu\r", cpuSpeed, gpuSpeed);
        }
        fflush(stdout);
        iterationsCount += 1;
    }
    while(now() - searchStartTime < options.timeout * 1000 and (not priorityQueue.isEmpty()));

    clearLine();
    printf("[RESULT] Solution: ");
    bestSolution->selectedValues.print(false);
    printf(" | Value: %u", bestSolution->cost);
    printf(" | Time: ");
    printElapsedTime(now() - searchStartTime);
    printf(" | Iterations: %u", iterationsCount);
    printf(" | States: %u - %u - %u\n", visitedStatesCount, priorityQueue.getSize(), filteredStatesCount);

    return EXIT_SUCCESS;
}

void configGPU()
{
    //Heap
    std::size_t sizeHeap = 3ul * 1024ul * 1024ul * 1024ul;
    hipDeviceSetLimit(hipLimitMallocHeapSize, sizeHeap);

    //Stack
    size_t sizeStackThread = 4 * 1024;
    hipDeviceSetLimit(hipLimitStackSize, sizeStackThread);

    //Cache
    hipDeviceSetCacheConfig(hipFuncCachePreferL1);
}

template<typename ProblemType, typename StateType>
void updatePriorityQueue(StateType* bestSolution, unsigned int * filteredStatesCount, PriorityQueue<StateType>* priorityQueue, OffloadBuffer<ProblemType, StateType>* offloadBuffer)
{
    for (unsigned int index = 0; index < offloadBuffer->getSize(); index += 1)
    {
        AugmentedState<StateType> const * parentAugmentedState = offloadBuffer->getAugmentedState(index);
        if(boundsCheck(bestSolution, parentAugmentedState))
        {
            Vector<StateType> const * const cutset = &offloadBuffer->getMDD(index)->cutset;
            for (StateType* cutsetState = cutset->begin(); cutsetState != cutset->end(); cutsetState += 1)
            {
                if (not priorityQueue->isFull())
                {
                    AugmentedState<StateType> const childAugmentedState(parentAugmentedState->upperbound, parentAugmentedState->lowerbound, cutsetState);
                    priorityQueue->insert(&childAugmentedState);
                }
            };
        }
        else
        {
            *filteredStatesCount += 1;
        }
    };
}

template<typename StateType>
bool boundsCheck(StateType* bestSolution, AugmentedState<StateType> const * augmentedState)
{
    return
       augmentedState->lowerbound < augmentedState->upperbound and
        augmentedState->lowerbound < bestSolution->cost and
        augmentedState->state->cost <= bestSolution->cost;
}

template<typename ProblemType, typename StateType>
bool checkForBetterSolutions(StateType* bestSolution, StateType* currentSolution, OffloadBuffer<ProblemType,StateType>* offloadBuffer)
{
    bool foundBetterSolution = false;

    for (unsigned int index = 0; index < offloadBuffer->getSize(); index += 1)
    {
        StateType const * const approximateSolution = &offloadBuffer->getMDD(index)->bottom;
        if (approximateSolution->cost < currentSolution->cost)
        {
            *currentSolution = *approximateSolution;
        }

        if (approximateSolution->cost < bestSolution->cost)
        {
            *bestSolution = *approximateSolution;
            foundBetterSolution = true;
        }
    };

    return foundBetterSolution;
}

template<typename ProblemType, typename StateType>
void prepareOffload(StateType* bestSolution, unsigned int * filteredStatesCount, PriorityQueue<StateType>* priorityQueue, OffloadBuffer<ProblemType, StateType>* offloadBuffer)
{
    offloadBuffer->clear();
    while (not (priorityQueue->isEmpty() or offloadBuffer->isFull()))
    {
        AugmentedState<StateType> const * const augmentedState = priorityQueue->front();
        if(boundsCheck(bestSolution, augmentedState))
        {
            offloadBuffer->enqueue(augmentedState);
        }
        else
        {
            *filteredStatesCount += 1;
        }
        priorityQueue->popFront();
    }
}

template<typename ProblemType, typename StateType>
void prepareOffload(AugmentedState<StateType> const * augmentedState, OffloadBuffer<ProblemType,StateType>* offloadBuffer)
{
    offloadBuffer->clear();
    while (not offloadBuffer->isFull())
    {
        offloadBuffer->enqueue(augmentedState);
    }
}

template<typename ProblemType, typename StateType>
void doOffloadCpuAsync(OffloadBuffer<ProblemType,StateType>* cpuOffloadBuffer, Vector<std::thread>* cpuThreads, bool onlyRestricted)
{
    cpuThreads->clear();
    for (unsigned int index = 0; index < cpuOffloadBuffer->getSize(); index += 1)
    {
        cpuThreads->resize(cpuThreads->getSize() + 1);
        new (cpuThreads->back()) std::thread(&OffloadBuffer<ProblemType,StateType>::doOffload, cpuOffloadBuffer, index, onlyRestricted);
    }
}

template<typename ProblemType, typename StateType>
void doOffloadGpuAsync(OffloadBuffer<ProblemType,StateType>* gpuOffloadBuffer, bool onlyRestricted)
{
    if(not gpuOffloadBuffer->isEmpty())
    {
        DD::MDD<ProblemType,StateType> const * const mdd = gpuOffloadBuffer->getMDD(0);
        unsigned int const blocksCount = gpuOffloadBuffer->getSize();
        unsigned int const blockSize = mdd->width * mdd->problem->maxBranchingFactor;
        assert(blockSize <= 1024);
        doOffloadKernel<ProblemType, StateType><<<blocksCount, blockSize, mdd->sizeOfScratchpadMemory()>>>(gpuOffloadBuffer, onlyRestricted);
    }
}

template<typename ProblemType, typename StateType>
void doOffloadsAsync(OffloadBuffer<ProblemType,StateType>* cpuOffloadBuffer, Vector<std::thread>* cpuThreads, OffloadBuffer<ProblemType,StateType>* gpuOffloadBuffer, bool onlyRestricted)
{
    doOffloadCpuAsync(cpuOffloadBuffer, cpuThreads, onlyRestricted);
    doOffloadGpuAsync(gpuOffloadBuffer, onlyRestricted);
}

template<typename ProblemType, typename StateType>
__global__
void doOffloadKernel(OffloadBuffer<ProblemType,StateType>* offloadBuffer, bool onlyRestricted)
{
    offloadBuffer->doOffload(blockIdx.x, onlyRestricted);
}

void waitOffloadCpu(Vector<std::thread>* cpuThreads, uint64_t* cpuOffloadEndTime)
{
    for (std::thread* thread = cpuThreads->begin(); thread != cpuThreads->end(); thread += 1)
    {
        if(thread->joinable())
        {
            thread->join();
        }
    }
    *cpuOffloadEndTime = now();
}

void waitOffloadGpu(uint64_t* gpuOffloadEndTime)
{
    hipDeviceSynchronize();
    *gpuOffloadEndTime = now();
}

void waitOffloads(Vector<std::thread>* cpuThreads, uint64_t* cpuOffloadEndTime, uint64_t* gpuOffloadEndTime)
{
    std::thread waitCpu(waitOffloadCpu, cpuThreads, cpuOffloadEndTime);
    std::thread waitGpu(waitOffloadGpu, gpuOffloadEndTime);

    waitCpu.join();
    waitGpu.join();
}

void printElapsedTime(uint64_t elapsedTimeMs)
{
    unsigned int ms = elapsedTimeMs;

    unsigned int h = ms / (1000 * 60 * 60);
    ms -= h * 1000 * 60 * 60;

    unsigned int m = ms / (1000 * 60);
    ms -= m * 1000 * 60;

    unsigned int s = ms / 1000;

    printf("%lums (%02uh%02um%02us)", elapsedTimeMs, h, m, s);
}

void clearLine()
{
    // ANSI clear line escape code
    printf("\33[2K\r");
}
