#include "hip/hip_runtime.h"
#include <algorithm>
#include <thread>
#include <Utils/Chrono.cuh>
#include "DP/VRPModel.cuh"
#include "OffloadBuffer.cuh"
#include "BB/PriorityQueue.cuh"

using namespace std;
using namespace Memory;
using namespace Chrono;
using namespace BB;
using namespace DD;
using namespace DP;
using namespace OP;
using ProblemType = VRProblem;
using StateType = VRPState;

// Auxiliary functions
void configGPU();

// Search
template<typename ProblemType, typename StateType>
void updatePriorityQueue(StateType* bestSolution, unsigned int * filteredStatesCount, PriorityQueue<StateType>* priorityQueue, OffloadBuffer<ProblemType, StateType>* offloadBuffer);

template<typename StateType>
bool boundsCheck(StateType* bestSolution, AugmentedState<StateType> const * augmentedState);

template<typename ProblemType, typename StateType>
bool checkForBetterSolutions(StateType* bestSolution, StateType* currentSolution, OffloadBuffer<ProblemType,StateType>* offloadBuffer);

// Offload
template<typename ProblemType, typename StateType>
void prepareOffload(StateType* bestSolution, unsigned int * filteredStatesCount, PriorityQueue<StateType>* priorityQueue, OffloadBuffer<ProblemType, StateType>* offloadBuffer);

template<typename ProblemType, typename StateType>
void prepareOffload(AugmentedState<StateType> const * augmentedState, OffloadBuffer<ProblemType,StateType>* offloadBuffer);

template<typename ProblemType, typename StateType>
void doOffloadCpuAsync(OffloadBuffer<ProblemType,StateType>* cpuOffloadBuffer, Vector<std::thread>* cpuThreads, bool onlyRestricted);

template<typename ProblemType, typename StateType>
void doOffloadGpuAsync(OffloadBuffer<ProblemType,StateType>* gpuOffloadBuffer, bool onlyRestricted);

template<typename ProblemType, typename StateType>
__global__ void doOffloadKernel(OffloadBuffer<ProblemType,StateType>* offloadBuffer, bool onlyRestricted);

void waitOffloadCpu(Vector<std::thread>* cpuThreads);

void waitOffloadGpu();

// Debug
void printElapsedTime(uint64_t elapsedTimeMs);

void clearLine();

int main(int argc, char ** argv)
{
    // Input parsing
    char const * problemFileName = argv[1];
    unsigned int const queueMaxSize = std::stoi(argv[2]);
    unsigned int const timeoutSeconds = std::stoi(argv[3]);
    unsigned int const cpuMaxWidth = std::stoi(argv[4]);
    unsigned int const cpuMaxParallelism = std::stoi(argv[5]);
    unsigned int const gpuMaxWidth = std::stoi(argv[6]);
    unsigned int const gpuMaxParallelism = std::stoi(argv[7]);
    unsigned int const lnsEqPercentage = std::stoi(argv[8]);
    unsigned int const lnsNeqPercentage = std::stoi(argv[9]);
    unsigned int const randomSeed = std::stoi(argv[10]);
    assert(lnsEqPercentage + lnsNeqPercentage <= 100);

    // *******************
    // Data initialization
    // *******************

    // Context initialization
    std::mt19937 rng(randomSeed);
    MallocType gpuMallocType = MallocType::Std;
    if (gpuMaxParallelism > 0)
    {
        gpuMallocType = MallocType::Managed;
        configGPU();
    };
    Vector<std::thread>* cpuThreads = new Vector<std::thread>(cpuMaxParallelism, MallocType::Std);

    // Problems
    ProblemType* const cpuProblem = VRProblem::parseGrubHubInstance(problemFileName, MallocType::Std);
    ProblemType* const gpuProblem = VRProblem::parseGrubHubInstance(problemFileName, gpuMallocType);

    // PriorityQueue
    PriorityQueue<StateType> priorityQueue(cpuProblem, queueMaxSize);

    // Offload
    unsigned int memorySize = sizeof(OffloadBuffer<ProblemType,StateType>);
    byte* memory = safeMalloc(memorySize, MallocType::Std);
    OffloadBuffer<ProblemType,StateType>* cpuOffloadBuffer = new (memory) OffloadBuffer<ProblemType,StateType>(cpuProblem, cpuMaxWidth, cpuMaxParallelism, MallocType::Std);
    memory = safeMalloc(memorySize, gpuMallocType);
    OffloadBuffer<ProblemType,StateType>* gpuOffloadBuffer = new (memory) OffloadBuffer<ProblemType,StateType>(gpuProblem, gpuMaxWidth, gpuMaxParallelism, gpuMallocType);

    // Solutions
    memorySize = sizeof(StateType);
    memory = safeMalloc(memorySize, MallocType::Std);
    StateType* bestSolution = new (memory) StateType(cpuProblem, MallocType::Std);
    bestSolution->cost = DP::MaxCost;
    memory = safeMalloc(memorySize, MallocType::Std);
    StateType* currentSolution = new (memory) StateType(cpuProblem, MallocType::Std);
    currentSolution->cost = DP::MaxCost;

    // Root
    memorySize = sizeof(StateType);
    memory = safeMalloc(memorySize, MallocType::Std);
    StateType* root = new (memory) StateType(cpuProblem, MallocType::Std);
    makeRoot(cpuProblem, root);
    AugmentedState<StateType> const augmentedRoot(DP::MaxCost, 0, root);
    priorityQueue.insert(&augmentedRoot);

    // Search
    unsigned int iterationsCount = 0;
    enum SearchStatus {BB, LNS} searchStatus = SearchStatus::BB;
    unsigned int visitedStatesCount = 0;
    unsigned int filteredStatesCount = 0;

    // ************
    // Begin search
    // ************
    clearLine();
    printf("[INFO] Start branch and bound search\n");
    uint64_t searchStartTime = now();
    do
    {
        switch(searchStatus)
        {
            case SearchStatus::BB:
            {
                if (priorityQueue.isFull())
                {
                    searchStatus = SearchStatus::LNS;
                    clearLine();
                    printf("[INFO] Switching to large neighborhood search\n");
                }

                prepareOffload(bestSolution, &filteredStatesCount, &priorityQueue, cpuOffloadBuffer);
                prepareOffload(bestSolution, &filteredStatesCount, &priorityQueue, gpuOffloadBuffer);
            }
                break;
            case SearchStatus::LNS:
            {
                prepareOffload(&augmentedRoot, cpuOffloadBuffer);
                prepareOffload(&augmentedRoot, gpuOffloadBuffer);
                cpuOffloadBuffer->generateNeighbourhoods(currentSolution, lnsEqPercentage, lnsNeqPercentage, &rng);
                gpuOffloadBuffer->generateNeighbourhoods(currentSolution, lnsEqPercentage, lnsNeqPercentage, &rng);
                currentSolution->setInvalid();
            }
                break;
        }

        uint64_t cpuOffloadStartTime = now();
        doOffloadCpuAsync(cpuOffloadBuffer, cpuThreads, searchStatus == SearchStatus::LNS);

        uint64_t gpuOffloadStartTime = now();
        doOffloadGpuAsync(gpuOffloadBuffer, searchStatus == SearchStatus::LNS);

        waitOffloadCpu(cpuThreads);
        waitOffloadGpu();

        visitedStatesCount += cpuOffloadBuffer->getSize();
        visitedStatesCount += gpuOffloadBuffer->getSize();

        bool foundBetterSolution =
                checkForBetterSolutions(bestSolution, currentSolution, cpuOffloadBuffer) or
                checkForBetterSolutions(bestSolution, currentSolution, gpuOffloadBuffer);

        updatePriorityQueue(bestSolution, &filteredStatesCount, &priorityQueue, cpuOffloadBuffer);
        updatePriorityQueue(bestSolution, &filteredStatesCount, &priorityQueue, gpuOffloadBuffer);

        if(foundBetterSolution)
        {
            clearLine();
            printf("[INFO] Better solution found: ");
            bestSolution->selectedValues.print(false);
            printf(" | Value: %u", bestSolution->cost);
            printf(" | Time: ");
            printElapsedTime(now() - searchStartTime);
            printf(" | Iterations: %u", iterationsCount);
            printf(" | States: %u - %u - %u\n", visitedStatesCount, priorityQueue.getSize(), filteredStatesCount);
        }
        else
        {
            unsigned long int cpuSpeed = 0;
            if (cpuOffloadBuffer->getSize() > 0)
            {
                uint64_t cpuOffloadElapsedTime = max(1ul, now() - cpuOffloadStartTime);
                cpuSpeed = cpuOffloadBuffer->getSize() * 1000 / cpuOffloadElapsedTime;
            }

            unsigned long int gpuSpeed = 0;
            if (gpuOffloadBuffer->getSize() > 0)
            {
                uint64_t gpuOffloadElapsedTime = max(1ul, now() - gpuOffloadStartTime);
                gpuSpeed = gpuOffloadBuffer->getSize() * 1000 / gpuOffloadElapsedTime;
            }
            printf("[INFO] Solution: ");
            currentSolution->selectedValues.print(false);
            printf(" | Value: %u", currentSolution->cost);
            printf(" | Time: ");
            printElapsedTime(now() - searchStartTime);
            printf(" | Iteration: %u", iterationsCount);
            printf(" | States: %u - %u - %u", visitedStatesCount, priorityQueue.getSize(), filteredStatesCount);
            printf(" | Speed: %lu - %lu\r", cpuSpeed, gpuSpeed);
        }
        fflush(stdout);
        iterationsCount += 1;
    }
    while(now() - searchStartTime < timeoutSeconds * 1000 and (not priorityQueue.isEmpty()));

    clearLine();
    printf("[RESULT] Solution: ");
    bestSolution->selectedValues.print(false);
    printf(" | Value: %u", bestSolution->cost);
    printf(" | Time: ");
    printElapsedTime(now() - searchStartTime);
    printf(" | Iterations: %u", iterationsCount);
    printf(" | States: %u - %u - %u\n", visitedStatesCount, priorityQueue.getSize(), filteredStatesCount);

    return EXIT_SUCCESS;
}

void configGPU()
{
    //Heap
    std::size_t sizeHeap = 3ul * 1024ul * 1024ul * 1024ul;
    hipDeviceSetLimit(hipLimitMallocHeapSize, sizeHeap);

    //Stack
    size_t sizeStackThread = 4 * 1024;
    hipDeviceSetLimit(hipLimitStackSize, sizeStackThread);

    //Cache
    hipDeviceSetCacheConfig(hipFuncCachePreferL1);
}

template<typename ProblemType, typename StateType>
void updatePriorityQueue(StateType* bestSolution, unsigned int * filteredStatesCount, PriorityQueue<StateType>* priorityQueue, OffloadBuffer<ProblemType, StateType>* offloadBuffer)
{
    for (unsigned int index = 0; index < offloadBuffer->getSize(); index += 1)
    {
        AugmentedState<StateType> const * parentAugmentedState = offloadBuffer->getAugmentedState(index);
        if(boundsCheck(bestSolution, parentAugmentedState))
        {
            Vector<StateType> const* const cutset = offloadBuffer->getMDD(index)->getCutset();
            for (StateType* cutsetState = cutset->begin(); cutsetState != cutset->end(); cutsetState += 1)
            {
                if (not priorityQueue->isFull())
                {
                    AugmentedState<StateType> const childAugmentedState(parentAugmentedState->upperbound, parentAugmentedState->lowerbound, cutsetState);
                    priorityQueue->insert(&childAugmentedState);
                }
            };
        }
        else
        {
            *filteredStatesCount += 1;
        }
    };
}

template<typename StateType>
bool boundsCheck(StateType* bestSolution, AugmentedState<StateType> const * augmentedState)
{
    return
       augmentedState->lowerbound < augmentedState->upperbound and
        augmentedState->lowerbound < bestSolution->cost and
        augmentedState->state->cost <= bestSolution->cost;
}

template<typename ProblemType, typename StateType>
bool checkForBetterSolutions(StateType* bestSolution, StateType* currentSolution, OffloadBuffer<ProblemType,StateType>* offloadBuffer)
{
    bool foundBetterSolution = false;

    for (unsigned int index = 0; index < offloadBuffer->getSize(); index += 1)
    {
        StateType const * const approximateSolution = offloadBuffer->getMDD(index)->getBottom();
        if (approximateSolution->cost < currentSolution->cost)
        {
            *currentSolution = *approximateSolution;
        }

        if (approximateSolution->cost < bestSolution->cost)
        {
            *bestSolution = *approximateSolution;
            foundBetterSolution = true;
        }
    };

    return foundBetterSolution;
}

template<typename ProblemType, typename StateType>
void prepareOffload(StateType* bestSolution, unsigned int * filteredStatesCount, PriorityQueue<StateType>* priorityQueue, OffloadBuffer<ProblemType, StateType>* offloadBuffer)
{
    offloadBuffer->clear();
    while (not (priorityQueue->isEmpty() or offloadBuffer->isFull()))
    {
        AugmentedState<StateType> const * const augmentedState = priorityQueue->front();
        if(boundsCheck(bestSolution, augmentedState))
        {
            offloadBuffer->enqueue(augmentedState);
        }
        else
        {
            *filteredStatesCount += 1;
        }
        priorityQueue->popFront();
    }
}

template<typename ProblemType, typename StateType>
void prepareOffload(AugmentedState<StateType> const * augmentedState, OffloadBuffer<ProblemType,StateType>* offloadBuffer)
{
    offloadBuffer->clear();
    while (not offloadBuffer->isFull())
    {
        offloadBuffer->enqueue(augmentedState);
    }
}


template<typename ProblemType, typename StateType>
void doOffloadCpuAsync(OffloadBuffer<ProblemType,StateType>* cpuOffloadBuffer, Vector<std::thread>* cpuThreads, bool onlyRestricted)
{
    cpuThreads->clear();
    for (unsigned int index = 0; index < cpuOffloadBuffer->getSize(); index += 1)
    {
        cpuThreads->resize(cpuThreads->getSize() + 1);
        new (cpuThreads->back()) std::thread(&OffloadBuffer<ProblemType,StateType>::doOffload, cpuOffloadBuffer, index, onlyRestricted);
    }
}

template<typename ProblemType, typename StateType>
void doOffloadGpuAsync(OffloadBuffer<ProblemType,StateType>* gpuOffloadBuffer, bool onlyRestricted)
{
    if(not gpuOffloadBuffer->isEmpty())
    {
        DD::MDD<ProblemType,StateType> const * const mdd = gpuOffloadBuffer->getMDD(0);
        unsigned int const blocksCount = gpuOffloadBuffer->getSize();
        unsigned int const blockSize = mdd->width * mdd->problem->maxBranchingFactor;
        assert(blockSize <= 1024);
        doOffloadKernel<ProblemType, StateType><<<blocksCount, blockSize>>>(gpuOffloadBuffer, onlyRestricted);
    }
}

template<typename ProblemType, typename StateType>
__global__
void doOffloadKernel(OffloadBuffer<ProblemType,StateType>* offloadBuffer, bool onlyRestricted)
{
    offloadBuffer->doOffload(blockIdx.x, onlyRestricted);
}

void waitOffloadCpu(Vector<std::thread>* cpuThreads)
{
    for (std::thread* thread = cpuThreads->begin(); thread != cpuThreads->end(); thread += 1)
    {
        if(thread->joinable())
        {
            thread->join();
        }
    }
}

void waitOffloadGpu()
{
    hipDeviceSynchronize();
}


void printElapsedTime(uint64_t elapsedTimeMs)
{
    unsigned int ms = elapsedTimeMs;

    unsigned int h = ms / (1000 * 60 * 60);
    ms -= h * 1000 * 60 * 60;

    unsigned int m = ms / (1000 * 60);
    ms -= m * 1000 * 60;

    unsigned int s = ms / 1000;

    printf("%lums (%02uh%02um%02us)", elapsedTimeMs, h, m, s);
}

void clearLine()
{
    // ANSI clear line escape code
    printf("\33[2K\r");
}
