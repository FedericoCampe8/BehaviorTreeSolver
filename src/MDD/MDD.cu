#include "hip/hip_runtime.h"
#include <cstdint>
#include <cmath>
#include <numeric>
#include <algorithm>

#include <thrust/transform_reduce.h>
#include <thrust/binary_search.h>
#include <thrust/functional.h>
#include <thrust/execution_policy.h>
#include <thrust/uninitialized_fill.h>

#include <Containers/RuntimeArray.cuh>
#include <Utils/Memory.cuh>

#include "MDD.cuh"

__host__ __device__
unsigned int MDD::calcFanout(OP::TSPProblem const * problem)
{
    unsigned int fanout = 0;

    for(OP::Variable * var = problem->vars.begin(); var != problem->vars.end(); var += 1)
    {
        fanout = max(OP::Variable::cardinality(*var), fanout);
    }
    return fanout;
}

__device__
void MDD::buildMddTopDown(OP::TSPProblem const* problem, unsigned int maxWidth, MDDType type, DP::TSPState& top, unsigned int cutsetMaxSize, unsigned int& cutsetSize, DP::TSPState* cutset, DP::TSPState& bottom, std::byte* scratchpad)
{
    //Current states buffer
    RuntimeArray<DP::TSPState> currentStatesBuffer(maxWidth, scratchpad);
    unsigned int stateStorageSize = DP::TSPState::sizeOfStorage(problem);
    RuntimeArray<std::byte> currentStatesStorages(stateStorageSize * maxWidth,  Memory::align(4, currentStatesBuffer.storageEnd()));
    for(unsigned int currentStateIdx = 0; currentStateIdx < currentStatesBuffer.getCapacity(); currentStateIdx += 1)
    {
        new (&currentStatesBuffer[currentStateIdx]) DP::TSPState(problem, &currentStatesStorages[stateStorageSize * currentStateIdx]);
    };

    //Next states buffer
    RuntimeArray<DP::TSPState> nextStatesBuffer(maxWidth, Memory::align(4, currentStatesStorages.storageEnd()));
    RuntimeArray<std::byte> nextStatesStorage(stateStorageSize * maxWidth, Memory::align(4, nextStatesBuffer.storageEnd()));
    for(unsigned int nextStateIdx = 0; nextStateIdx < nextStatesBuffer.getCapacity(); nextStateIdx += 1)
    {
        new (&nextStatesBuffer[nextStateIdx]) DP::TSPState(problem, &nextStatesStorage[stateStorageSize * nextStateIdx]);
    };

    //Auxiliary information
    unsigned int fanout = cutsetMaxSize / maxWidth;
    RuntimeArray<int32_t> costs(fanout * maxWidth, Memory::align(4, nextStatesStorage.storageEnd()));
    RuntimeArray<uint32_t> indices(fanout * maxWidth, Memory::align(4, costs.storageEnd()));

    //Root
    currentStatesBuffer[0] = top;

    //Build
    bool cutsetInitialized = false;
    unsigned int currentStatesCount = 1;
    unsigned int nextStatesCount = 0;
    for(unsigned int level = top.selectedValues.getSize(); level < problem->vars.getCapacity(); level += 1)
    {
        //Initialize indices
        for(unsigned int i = 0; i < indices.getCapacity(); i += 1)
        {
            indices.at(i) = i;
        }

        //Initialize costs
        for(unsigned int i = 0; i < costs.getCapacity(); i +=1)
        {
            costs.at(i) = INT32_MAX;
        }

        //Calculate costs
        assert(currentStatesCount <= currentStatesBuffer.getCapacity());
        for(unsigned int currentStateIdx = 0; currentStateIdx < currentStatesCount; currentStateIdx += 1)
        {
            DP::TSPModel::calcCosts(problem, level, &currentStatesBuffer[currentStateIdx], &costs[fanout * currentStateIdx]);
        }

        //Sort indices by costs
        thrust::sort_by_key(thrust::seq, costs.begin(), costs.end(), indices.begin());

        //Count next states
        int* costsEnd = thrust::lower_bound(thrust::seq, costs.begin(), costs.end(), INT32_MAX);
        unsigned int costsCount = thrust::distance(costs.begin(), costsEnd);

        nextStatesCount = min(maxWidth, costsCount);
        nextStatesCount = level < problem->vars.getCapacity() - 1 ? nextStatesCount : 1;

        //Cutset
        if(costsCount > maxWidth and type == MDDType::Relaxed and (not cutsetInitialized))
        {
            thrust::for_each(thrust::seq, indices.begin(), indices.begin() + costsCount, [=] (unsigned int& index)
            {
                unsigned int currentStateIdx = index / fanout;
                unsigned int edgeIdx = index % fanout;
                int value = problem->vars[level].minValue + edgeIdx;
                unsigned int cutsetStateIdx = thrust::distance(indices.begin(), &index);
                DP::TSPModel::makeNextState(problem, &currentStatesBuffer[currentStateIdx], value, costs[cutsetStateIdx], &cutset[cutsetStateIdx]);
            });

            cutsetSize = costsCount;
            cutsetInitialized = true;
        }

        //Add states
        assert(nextStatesCount <= indices.getCapacity());
        thrust::for_each(thrust::seq, indices.begin(), indices.begin() + costsCount, [=] (unsigned int& index)
        {
            unsigned int currentStateIdx = index / fanout;
            unsigned int edgeIdx =  index % fanout;
            int value = problem->vars[level].minValue + edgeIdx;
            unsigned int nextStateIdx = thrust::distance(indices.begin(), &index);
            if(nextStateIdx < nextStatesCount)
            {
                DP::TSPModel::makeNextState(problem, &currentStatesBuffer[currentStateIdx], value, costs[nextStateIdx], &nextStatesBuffer[nextStateIdx]);
            }
            else if (type == Relaxed)
            {
                DP::TSPModel::mergeNextState(problem, &currentStatesBuffer[currentStateIdx], value, &nextStatesBuffer[nextStatesCount - 1]);
            }
        });

        //Prepare for the next loop
        currentStatesBuffer.swap(nextStatesBuffer);
        currentStatesCount = nextStatesCount;
        nextStatesCount = 0;
    }

    //Copy bottom
    bottom = currentStatesBuffer[0];
}